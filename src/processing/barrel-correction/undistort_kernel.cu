/**
 * W23: CUDA Barrel Distortion Correction Kernel
 * Target: 125 FPS @ 4056×3040 (IMX477 full resolution)
 *
 * Applies inverse barrel distortion using Brown-Conrady model:
 * x_corrected = x_distorted * (1 + k1*r² + k2*r⁴ + k3*r⁶)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * CUDA kernel for barrel distortion correction
 * Each thread processes one pixel
 */
__global__ void undistort_kernel(
    const unsigned char* __restrict__ input,
    unsigned char* __restrict__ output,
    int width, int height,
    float k1, float k2, float k3,  // Radial distortion coefficients
    float cx, float cy,             // Principal point (image center)
    float fx, float fy              // Focal length in pixels
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    // Normalize coordinates to [-1, 1] range
    float xn = (x - cx) / fx;
    float yn = (y - cy) / fy;

    // Calculate radial distance squared
    float r2 = xn*xn + yn*yn;
    float r4 = r2 * r2;
    float r6 = r4 * r2;

    // Apply Brown-Conrady distortion model (inverse)
    float radial_distortion = 1.0f + k1*r2 + k2*r4 + k3*r6;

    // Distort coordinates
    float xu = xn * radial_distortion;
    float yu = yn * radial_distortion;

    // Denormalize back to pixel coordinates
    int src_x = (int)(xu * fx + cx);
    int src_y = (int)(yu * fy + cy);

    // Bounds check with clamping
    src_x = max(0, min(src_x, width - 1));
    src_y = max(0, min(src_y, height - 1));

    // Copy RGB pixel (3 bytes per pixel)
    int dst_idx = (y * width + x) * 3;
    int src_idx = (src_y * width + src_x) * 3;

    output[dst_idx + 0] = input[src_idx + 0];  // R
    output[dst_idx + 1] = input[src_idx + 1];  // G
    output[dst_idx + 2] = input[src_idx + 2];  // B
}

/**
 * Host function to launch undistortion kernel
 * Processes a single RGB frame
 */
extern "C" {
    void undistort_frame(
        const unsigned char* h_input,
        unsigned char* h_output,
        int width, int height,
        float k1, float k2, float k3,
        float cx, float cy, float fx, float fy
    ) {
        // Allocate device memory
        unsigned char *d_input, *d_output;
        size_t image_size = width * height * 3;  // RGB

        hipMalloc(&d_input, image_size);
        hipMalloc(&d_output, image_size);

        // Copy input frame to device
        hipMemcpy(d_input, h_input, image_size, hipMemcpyHostToDevice);

        // Configure kernel launch parameters
        // 16x16 threads per block is optimal for most GPUs
        dim3 block(16, 16);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y
        );

        // Launch kernel
        undistort_kernel<<<grid, block>>>(
            d_input, d_output, width, height,
            k1, k2, k3, cx, cy, fx, fy
        );

        // Wait for kernel to complete
        hipDeviceSynchronize();

        // Copy result back to host
        hipMemcpy(h_output, d_output, image_size, hipMemcpyDeviceToHost);

        // Cleanup device memory
        hipFree(d_input);
        hipFree(d_output);
    }

    /**
     * Batch processing version for multiple frames
     * More efficient for video processing
     */
    void undistort_batch(
        const unsigned char** h_inputs,
        unsigned char** h_outputs,
        int num_frames,
        int width, int height,
        float k1, float k2, float k3,
        float cx, float cy, float fx, float fy
    ) {
        size_t frame_size = width * height * 3;

        // Allocate device memory for batch
        unsigned char *d_input, *d_output;
        hipMalloc(&d_input, frame_size * num_frames);
        hipMalloc(&d_output, frame_size * num_frames);

        // Copy all frames to device
        for (int i = 0; i < num_frames; i++) {
            hipMemcpy(
                d_input + i * frame_size,
                h_inputs[i],
                frame_size,
                hipMemcpyHostToDevice
            );
        }

        // Launch kernel for each frame
        dim3 block(16, 16);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y
        );

        for (int i = 0; i < num_frames; i++) {
            undistort_kernel<<<grid, block>>>(
                d_input + i * frame_size,
                d_output + i * frame_size,
                width, height,
                k1, k2, k3, cx, cy, fx, fy
            );
        }

        hipDeviceSynchronize();

        // Copy all frames back to host
        for (int i = 0; i < num_frames; i++) {
            hipMemcpy(
                h_outputs[i],
                d_output + i * frame_size,
                frame_size,
                hipMemcpyDeviceToHost
            );
        }

        // Cleanup
        hipFree(d_input);
        hipFree(d_output);
    }
}
